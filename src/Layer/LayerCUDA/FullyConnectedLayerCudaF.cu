#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <>
#include <stdlib.h>
#include <stdio.h>
#include <FLOAT.h>
#include "MkFn.h"
#include "../../Lib/Lib.h"
#include "LibCuda.h"
#include "FullyConnectedLayerCudaF.h"
__constant__ int _BatchSize;
__constant__ float _LearningRate;

void FullyConnectedLayerCudaF::GetFieldName(int field_idx, wchar_t* name){
	switch(field_idx){
		case 0: wcscpy(name, L"X"); break;
		case 1: wcscpy(name, L"Y"); break;
		case 2: wcscpy(name, L"x"); break;
		case 3: wcscpy(name, L"y"); break;
		case 4: wcscpy(name, L"w"); break;
		case 5: wcscpy(name, L"b"); break;
		case 6: wcscpy(name, L"u"); break;
		case 7: wcscpy(name, L"delta_x"); break;
		case 8: wcscpy(name, L"delta_y"); break;
		case 9: wcscpy(name, L"delta_w"); break;
		case 10: wcscpy(name, L"delta_b"); break;
		case 11: wcscpy(name, L"delta_u"); break;
		default: name[0] = 0; break;
	}
}

int FullyConnectedLayerCudaF::GetFieldDimension(int field_idx){
	switch(field_idx){
	case 0: return 0;
	case 1: return 0;
	case 2: return 1;
	case 3: return 1;
	case 4: return 2;
	case 5: return 1;
	case 6: return 1;
	case 7: return 1;
	case 8: return 1;
	case 9: return 2;
	case 10: return 1;
	case 11: return 1;
	default: return -1;
	}
}

int* FullyConnectedLayerCudaF::GetFieldSize(int field_idx){
	switch(field_idx){
	case 0: return 0;
	case 1: return 0;
	case 2:
		x_size_[0] = X;
		return x_size_;
	case 3:
		y_size_[0] = Y;
		return y_size_;
	case 4:
		w_size_[0] = Y;
		w_size_[1] = X;
		return w_size_;
	case 5:
		b_size_[0] = Y;
		return b_size_;
	case 6:
		u_size_[0] = Y;
		return u_size_;
	case 7:
		delta_x_size_[0] = X;
		return delta_x_size_;
	case 8:
		delta_y_size_[0] = Y;
		return delta_y_size_;
	case 9:
		delta_w_size_[0] = Y;
		delta_w_size_[1] = X;
		return delta_w_size_;
	case 10:
		delta_b_size_[0] = Y;
		return delta_b_size_;
	case 11:
		delta_u_size_[0] = Y;
		return delta_u_size_;
	default: return 0;
	}
}

void FullyConnectedLayerCudaF::GetFieldValue(int field_idx, void* dst){
	int _cnt = GetFieldElementCount(field_idx);
	switch(field_idx){
	case 0: memcpy(dst, &X, _cnt * sizeof(int)); break;
	case 1: memcpy(dst, &Y, _cnt * sizeof(int)); break;
	case 2: memcpy(dst, x, _cnt * sizeof(float)); break;
	case 3: memcpy(dst, y, _cnt * sizeof(float)); break;
	case 4: memcpy(dst, w, _cnt * sizeof(float)); break;
	case 5: memcpy(dst, b, _cnt * sizeof(float)); break;
	case 6: memcpy(dst, u, _cnt * sizeof(float)); break;
	case 7: memcpy(dst, delta_x, _cnt * sizeof(float)); break;
	case 8: memcpy(dst, delta_y, _cnt * sizeof(float)); break;
	case 9: memcpy(dst, delta_w, _cnt * sizeof(float)); break;
	case 10: memcpy(dst, delta_b, _cnt * sizeof(float)); break;
	case 11: memcpy(dst, delta_u, _cnt * sizeof(float)); break;
	}
}

void FullyConnectedLayerCudaF::SetFieldValue(int field_idx, void* src){
	int _cnt = GetFieldElementCount(field_idx);
	switch(field_idx){
	case 0: memcpy(&X, src, _cnt * sizeof(int)); break;
	case 1: memcpy(&Y, src, _cnt * sizeof(int)); break;
	case 2: memcpy(x, src, _cnt * sizeof(float)); break;
	case 3: memcpy(y, src, _cnt * sizeof(float)); break;
	case 4: memcpy(w, src, _cnt * sizeof(float)); break;
	case 5: memcpy(b, src, _cnt * sizeof(float)); break;
	case 6: memcpy(u, src, _cnt * sizeof(float)); break;
	case 7: memcpy(delta_x, src, _cnt * sizeof(float)); break;
	case 8: memcpy(delta_y, src, _cnt * sizeof(float)); break;
	case 9: memcpy(delta_w, src, _cnt * sizeof(float)); break;
	case 10: memcpy(delta_b, src, _cnt * sizeof(float)); break;
	case 11: memcpy(delta_u, src, _cnt * sizeof(float)); break;
	}
}

FullyConnectedLayerCudaF::FullyConnectedLayerCudaF(int x_size, int y_size){

    // X = x_size
    X = x_size;


    // Y = y_size
    Y = y_size;

	SetNormalRand(w, Y * X);
	SetNormalRand(b, Y);
	_chk(hipStreamCreate(&_stream_y));
	_chk(hipStreamCreate(&_stream_u));
	_chk(hipStreamCreate(&_stream_delta_x));
	_chk(hipStreamCreate(&_stream_delta_w));
	_chk(hipStreamCreate(&_stream_delta_b));
	_chk(hipStreamCreate(&_stream_delta_u));

	_chk(hipEventCreate(&_event_y));
	_chk(hipEventCreate(&_event_u));
	_chk(hipEventCreate(&_event_delta_x));
	_chk(hipEventCreate(&_event_delta_w));
	_chk(hipEventCreate(&_event_delta_b));
	_chk(hipEventCreate(&_event_delta_u));

}

FullyConnectedLayerCudaF::~FullyConnectedLayerCudaF(){
	Free();
	_chk(hipStreamDestroy(_stream_y));
	_chk(hipStreamDestroy(_stream_u));
	_chk(hipStreamDestroy(_stream_delta_x));
	_chk(hipStreamDestroy(_stream_delta_w));
	_chk(hipStreamDestroy(_stream_delta_b));
	_chk(hipStreamDestroy(_stream_delta_u));

	_chk(hipEventDestroy(_event_y));
	_chk(hipEventDestroy(_event_u));
	_chk(hipEventDestroy(_event_delta_x));
	_chk(hipEventDestroy(_event_delta_w));
	_chk(hipEventDestroy(_event_delta_b));
	_chk(hipEventDestroy(_event_delta_u));

	_Free(w);
	_Free(b);

}

void FullyConnectedLayerCudaF::Allocate(){
	_chk(_Malloc(y, BatchSize * Y * sizeof(float))); 
	_chk(_Malloc(u, BatchSize * Y * sizeof(float))); 
	_chk(_Malloc(delta_x, BatchSize * X * sizeof(float))); 
	_chk(_Malloc(delta_w, BatchSize * Y * X * sizeof(float))); 
	_chk(_Malloc(delta_b, BatchSize * Y * sizeof(float))); 
	_chk(_Malloc(delta_u, BatchSize * Y * sizeof(float))); 
}

void FullyConnectedLayerCudaF::Free(){
	_chk(_Free(y)); 
	_chk(_Free(u)); 
	_chk(_Free(delta_x)); 
	_chk(_Free(delta_w)); 
	_chk(_Free(delta_b)); 
	_chk(_Free(delta_u)); 
}
__global__ static void forward_u(float* u, int X, float* x, float* w, float* b, int Y){
	int _batch_idx = threadIdx.x;
	int i = blockIdx.x;

    // u[i] = (from j in Range(X) select x[j] * w[i, j]).Sum() + b[i]
    float _wk1 = 0;
    for(int j = 0; j < X; j++){
        _wk1 += x[(j) * _BatchSize + _batch_idx] * w[X * i + j];
    }
    u[(i) * _BatchSize + _batch_idx] = _wk1 + b[i];

}
void FullyConnectedLayerCudaF::Start_forward_u(){
	int blocks_x = 1;
	int blocks_y = 1;
	int blocks_z = 1;
	blocks_x = Y;
	dim3 threadsPerBlock = dim3(BatchSize);
	dim3 blocksPerGrid   = dim3(blocks_x, blocks_y, blocks_z);
	forward_u<<<blocksPerGrid, threadsPerBlock, 0, _stream_u>>>(u, X, x, w, b, Y);
	_chk(hipEventRecord(_event_u, _stream_u));
}
__global__ static void forward_y(float* y, float* u, int Y){
	int _batch_idx = threadIdx.x;
	int i = blockIdx.x;

    // y[i] = sigmoid(u[i])
    y[(i) * _BatchSize + _batch_idx] = sigmoid(u[(i) * _BatchSize + _batch_idx]);

}
void FullyConnectedLayerCudaF::Start_forward_y(){
	int blocks_x = 1;
	int blocks_y = 1;
	int blocks_z = 1;
	blocks_x = Y;
	dim3 threadsPerBlock = dim3(BatchSize);
	dim3 blocksPerGrid   = dim3(blocks_x, blocks_y, blocks_z);
	_chk(hipStreamWaitEvent(_stream_y, _event_u, 0));
	forward_y<<<blocksPerGrid, threadsPerBlock, 0, _stream_y>>>(y, u, Y);
	_chk(hipEventRecord(_event_y, _stream_y));
}
void FullyConnectedLayerCudaF::Forward(){
	_chk(_MemcpyToSymbol(_BatchSize, BatchSize, sizeof(BatchSize)));
	Start_forward_u();
	Start_forward_y();
}
__global__ static void backward_delta_u(float* delta_u, float* delta_y, float* u, int Y){
	int _batch_idx = threadIdx.x;
	int i = blockIdx.x;

    // delta_u[i] = delta_y[i] * sigmoid_prime(u[i])
    delta_u[(i) * _BatchSize + _batch_idx] = delta_y[(i) * _BatchSize + _batch_idx] * sigmoid_prime(u[(i) * _BatchSize + _batch_idx]);

}
void FullyConnectedLayerCudaF::Start_backward_delta_u(){
	int blocks_x = 1;
	int blocks_y = 1;
	int blocks_z = 1;
	blocks_x = Y;
	dim3 threadsPerBlock = dim3(BatchSize);
	dim3 blocksPerGrid   = dim3(blocks_x, blocks_y, blocks_z);
	backward_delta_u<<<blocksPerGrid, threadsPerBlock, 0, _stream_delta_u>>>(delta_u, delta_y, u, Y);
	_chk(hipEventRecord(_event_delta_u, _stream_delta_u));
}
__global__ static void backward_delta_x(float* delta_x, int Y, float* delta_u, float* w, int X){
	int _batch_idx = threadIdx.x;
	int i_j = blockIdx.x;

    // delta_x[i_j] = (from i in Range(Y) select delta_u[i] * w[i, i_j]).Sum()
    float _wk2 = 0;
    for(int i = 0; i < Y; i++){
        _wk2 += delta_u[(i) * _BatchSize + _batch_idx] * w[X * i + i_j];
    }
    delta_x[(i_j) * _BatchSize + _batch_idx] = _wk2;

}
void FullyConnectedLayerCudaF::Start_backward_delta_x(){
	int blocks_x = 1;
	int blocks_y = 1;
	int blocks_z = 1;
	blocks_x = X;
	dim3 threadsPerBlock = dim3(BatchSize);
	dim3 blocksPerGrid   = dim3(blocks_x, blocks_y, blocks_z);
	_chk(hipStreamWaitEvent(_stream_delta_x, _event_delta_u, 0));
	backward_delta_x<<<blocksPerGrid, threadsPerBlock, 0, _stream_delta_x>>>(delta_x, Y, delta_u, w, X);
	_chk(hipEventRecord(_event_delta_x, _stream_delta_x));
}
__global__ static void backward_delta_w(float* delta_w, float* delta_u, float* x, int Y, int X){
	int _batch_idx = threadIdx.x;
	int i = blockIdx.y;
	int i_j = blockIdx.x;

    // delta_w[i, i_j] = delta_u[i] * x[i_j]
    delta_w[(X * i + i_j) * _BatchSize + _batch_idx] = delta_u[(i) * _BatchSize + _batch_idx] * x[(i_j) * _BatchSize + _batch_idx];

}
void FullyConnectedLayerCudaF::Start_backward_delta_w(){
	int blocks_x = 1;
	int blocks_y = 1;
	int blocks_z = 1;
	blocks_y = Y;
	blocks_x = X;
	dim3 threadsPerBlock = dim3(BatchSize);
	dim3 blocksPerGrid   = dim3(blocks_x, blocks_y, blocks_z);
	_chk(hipStreamWaitEvent(_stream_delta_w, _event_delta_u, 0));
	backward_delta_w<<<blocksPerGrid, threadsPerBlock, 0, _stream_delta_w>>>(delta_w, delta_u, x, Y, X);
	_chk(hipEventRecord(_event_delta_w, _stream_delta_w));
}
__global__ static void backward_delta_b(float* delta_b, float* delta_u, int Y){
	int _batch_idx = threadIdx.x;
	int i = blockIdx.x;

    // delta_b[i] = delta_u[i]
    delta_b[(i) * _BatchSize + _batch_idx] = delta_u[(i) * _BatchSize + _batch_idx];

}
void FullyConnectedLayerCudaF::Start_backward_delta_b(){
	int blocks_x = 1;
	int blocks_y = 1;
	int blocks_z = 1;
	blocks_x = Y;
	dim3 threadsPerBlock = dim3(BatchSize);
	dim3 blocksPerGrid   = dim3(blocks_x, blocks_y, blocks_z);
	_chk(hipStreamWaitEvent(_stream_delta_b, _event_delta_u, 0));
	backward_delta_b<<<blocksPerGrid, threadsPerBlock, 0, _stream_delta_b>>>(delta_b, delta_u, Y);
	_chk(hipEventRecord(_event_delta_b, _stream_delta_b));
}
void FullyConnectedLayerCudaF::Backward(){
	_chk(_MemcpyToSymbol(_BatchSize, BatchSize, sizeof(BatchSize)));
	Start_backward_delta_u();
	Start_backward_delta_x();
	Start_backward_delta_w();
	Start_backward_delta_b();
}
__global__ static void UpdateParameterKernel_0(float* w, float* delta_w){
	int _idx = (blockIdx.x) * blockDim.x + threadIdx.x;
	int offset = _idx * _BatchSize;
	{
		float sum = 0;
		for (int i = 0; i < _BatchSize; i++) {
			sum += delta_w[offset + i];
		}
		w[_idx] -= _LearningRate * sum;
	}
}
void FullyConnectedLayerCudaF::UpdateParameter_0(){
	int threads_x = 1;
	int blocks_x = 1;
	int blocks_y = 1;
	int blocks_z = 1;
	blocks_x = Y;
	threads_x = X;
	dim3 threadsPerBlock = dim3(threads_x);
	dim3 blocksPerGrid   = dim3(blocks_x, blocks_y, blocks_z);
	UpdateParameterKernel_0<<<blocksPerGrid, threadsPerBlock>>>(w, delta_w);
}
__global__ static void UpdateParameterKernel_1(float* b, float* delta_b){
	int _idx = threadIdx.x;
	int offset = _idx * _BatchSize;
	{
		float sum = 0;
		for (int i = 0; i < _BatchSize; i++) {
			sum += delta_b[offset + i];
		}
		b[_idx] -= _LearningRate * sum;
	}
}
void FullyConnectedLayerCudaF::UpdateParameter_1(){
	int threads_x = 1;
	int blocks_x = 1;
	int blocks_y = 1;
	int blocks_z = 1;
	threads_x = Y;
	dim3 threadsPerBlock = dim3(threads_x);
	dim3 blocksPerGrid   = dim3(blocks_x, blocks_y, blocks_z);
	UpdateParameterKernel_1<<<blocksPerGrid, threadsPerBlock>>>(b, delta_b);
}
void FullyConnectedLayerCudaF::UpdateParameter(){
	_chk(_MemcpyToSymbol(_BatchSize, BatchSize, sizeof(BatchSize)));
	_chk(_MemcpyToSymbol(_LearningRate, LearningRate, sizeof(LearningRate)));
	_chk(hipDeviceSynchronize());
	UpdateParameter_0();
	UpdateParameter_1();
	_chk(hipDeviceSynchronize());
}

extern "C" DllExport Layer* MakeFullyConnectedLayerCudaF(int x_size, int y_size){
	return new FullyConnectedLayerCudaF(x_size, y_size);
}
